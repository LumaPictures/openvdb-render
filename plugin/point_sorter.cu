// Copyright 2019 Luma Pictures
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "point_sorter.h"

#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

bool cuda_available() {
    int device_count = 0;
    hipGetDeviceCount(&device_count);

    std::cerr << "[openvdb_render] Checking for CUDA support!" << std::endl;
    for(int i = 0; i < device_count; ++i)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cerr << "\tFound CUDA device : " << prop.name << std::endl;
    }

    return device_count > 0;
}

struct sort_points_functor
{
    float camera_pos[3];

    sort_points_functor(const float* cp) {
        camera_pos[0] = cp[0];
        camera_pos[1] = cp[1];
        camera_pos[2] = cp[2];
    }

    __host__ __device__
    bool operator()(PointData x, PointData y)
    {
        float rx = x.pos[0] - camera_pos[0];
        rx = rx * rx;
        float t = x.pos[1] - camera_pos[1];
        rx += t * t;
        t = x.pos[2] - camera_pos[2];
        rx += t * t;
        float ry = y.pos[0] - camera_pos[0];
        ry = ry * ry;
        t = y.pos[1] - camera_pos[1];
        ry += t * t;
        t = y.pos[2] - camera_pos[2];
        ry += t * t;
        return rx > ry;
    }
};

void sort_points(PointData* data, size_t point_count, const float* camera_position) {
    thrust::device_vector<PointData> device_vector(point_count);
    thrust::copy(data, data + point_count, device_vector.begin());
    thrust::sort(device_vector.begin(), device_vector.end(), sort_points_functor(camera_position));
    thrust::copy(device_vector.begin(), device_vector.end(), data);
}
