#include "point_sorter.h"

#include <iostream>

bool cuda_available() {
    int device_count = 0;
    hipGetDeviceCount(&device_count);

    std::cerr << "[openvdb_render] Checking for CUDA support!" << std::endl;
    for(int i = 0; i < device_count; ++i)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cerr << "\t\tFound CUDA device : " << prop.name << std::endl;
    }

    return device_count > 0;
}

void sort_points(PointData* data, size_t point_count) {

}
